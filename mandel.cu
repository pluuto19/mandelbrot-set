#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define IMG_W 1920
#define IMG_H 1080
#define CHANNELS 3
#define MAX_ITR 100
#define REAL_MIN -2.0
#define REAL_MAX 1.0
#define IMAG_MIN -1.5
#define IMAG_MAX 1.5
#define INC_REAL (REAL_MAX - REAL_MIN) / IMG_W
#define INC_IMAG (IMAG_MAX - IMAG_MIN) / IMG_H
#define NUM_THREADS 32 * 32 // GTX 1650 supports threads launching in 2 dimnesions, each with 32 threads
#define NUM_BLOCKS (int)ceil(((float)IMG_W * IMG_H) / NUM_THREADS)

typedef struct complexNumber
{
    float real;
    float imag;
} C;

float complexAbs(C *c)
{
    return sqrt((c->real * c->real) + (c->imag * c->imag));
}

void complexAdd(C *z, C *cnst, C *res)
{
    res->real = z->real + cnst->real;
    res->imag = z->imag + cnst->imag;
}

void complexMult(C *x, C *y, C *res)
{
    res->real = (x->real * y->real) - (x->imag * y->imag);
    res->imag = (x->real * y->imag) + (x->imag * y->real);
}

int mandelbrot(C *cnst)
{
    C z = {0.0, 0.0};
    C zSq;
    for (int i = 0; i < MAX_ITR; i++)
    {
        if (complexAbs(&z) > 2)
        {
            return i;
        }
        complexMult(&z, &z, &zSq);
        complexAdd(&zSq, cnst, &z);
    }
    return MAX_ITR;
}

void getColor(int itrs, unsigned char *r, unsigned char *g, unsigned char *b)
{
    *r = (unsigned char)(itrs * 2.0f);
    *g = (unsigned char)(itrs * 1.9f);
    *b = (unsigned char)(itrs * 2.35f);
}

__global__ void parallelMandelbrot(unsigned char *dev_image){
    //int x = interpolate threadIdx.x and threadIdx.y and block
    //int y = 
    // if x*y < N
}

int main(void){
    // 1D blocks with 2D threads
    // I would have to interpolate blockIdx and threadIdx in x and y to find x and y pixels co-ords
    // formula to calculate real and imag for each parallel kernel
    // real = real_min + (x * real_inc) , x is interpolated
    // imag = imag_min + (y * imag_inc) , y is interpolated
    // each kernel will be launched with a zoom factor as function parameter
    // flow:
    // loop with scaling factor
    // launch kernel with scaling factor
    // inside kernel:
    // calculate x and y by interpolating
    // calculate real and imag
    // run function normally
    // kernel end
    // cudadevice synch
    // devicetohostcopy
    // pass to stb_img
    // repeat loop
    unsigned char *host_image = (unsigned char *)malloc(IMG_W * IMG_H * CHANNELS); //allocate memory on host before launching kernel to save computations in case memory fails to allocate
    
    if (host_image == NULL)
    {
        fprintf(stderr, "Failed to allocate memory\n");
        return 1;
    }

    unsigned char *dev_image;

    hipMalloc(&dev_image, IMG_W * IMG_H * CHANNELS);

    // handle allocation error

    dim3 block_dim(32,32,1);

    parallelMandelbrot<<<NUM_BLOCKS, block_dim>>>(dev_image);

    hipMemcpy(host_image, dev_image, IMG_W * IMG_H * CHANNELS, hipMemcpyDeviceToHost);

    hipFree(dev_image);

    if (!stbi_write_png("mandel-c.png", IMG_W, IMG_H, CHANNELS, host_image, IMG_W * CHANNELS))
    {
        fprintf(stderr, "Failed to write image\n");
        return 1;
    }

    free(host_image);
    printf("Image written to mandel-c.png\n");
}
